#include "hip/hip_runtime.h"
// #include <cstdlib>

// #include <chrono>
// #include <iostream>

// #include "hip/hip_runtime.h"
// #include ""

// void Add(int n, float* x, float* y)
// {
//     int index  = threadIdx.x;
//     int stride = blockDim.x;

//     for (int i = index; i < n; i += stride)
//     {
//         y[i] += x[i];
//         std::cout << "\rScanlines remaining: " << i << ' ' << std::flush;
//     }
// }

// __global__ void GPU_Add(int n, float* x, float* y)
// {
//     for (int i = 0; i < n; i++)
//     {
//         y[i] += x[i];
//         // std::cout << "\rScanlines remaining: " << i << ' ' << std::flush;
//     }
// }

// int main(int argc, char** argv)
// {
//     constexpr int N = 1 << 25;

//     // auto *x = new float[N], *y = new float[N];

//     // 分配CUDA内存
//     float *   x, *y;
//     hipError_t err = hipMallocManaged(&x, N * sizeof(float));
//     err           = hipMallocManaged(&y, N * sizeof(float));

//     for (int i = 0; i < N; i++)
//     {
//         x[i] = 1.0f;
//         y[i] = 2.0f;
//     }

//     // 计时
//     auto start = std::chrono::system_clock::now();

//     // Add(N, x, y);
//     // 启动CUDA核函数
//     GPU_Add<<<1, 256>>>(N, x, y);
//     std::cout << "GPU_Add<<<1, 1>>>(N, x, y);" << std::endl;
//     hipDeviceSynchronize();
//     std::cout << "hipDeviceSynchronize();" << std::endl;

//     // 计时
//     auto end       = std::chrono::system_clock::now();
//     using timeType = std::chrono::milliseconds;
//     auto duration  = std::chrono::duration_cast<timeType>(end - start);
//     std::cout << "Time: " << duration.count() << "ms" << std::endl;

//     float maxError = 0.0f;
//     for (int i = 0; i < N; i++)
//     {
//         maxError = fmax(maxError, fabs(y[i] - 3.0f));
//     }

//     std::cout << "Max error: " << maxError << std::endl;

//     // delete[] x, y;
//     hipFree(x);
//     hipFree(y);

//     return EXIT_SUCCESS;
// }

#include <cstdio>
#include <cstdlib>

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

template <typename T> __global__ void matAdd_cuda(T* a, T* b, T* sum)
{
    int i  = blockIdx.x * blockDim.x + threadIdx.x;
    sum[i] = a[i] + b[i];
}

float* matAdd(float* a, float* b, int length)
{
    int device = 0;  // 设置使用第0块GPU进行运算
    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    int    threadMaxSize = deviceProp.maxThreadsPerBlock;
    int    blockSize     = (length + threadMaxSize - 1) / threadMaxSize;
    dim3   thread(threadMaxSize), block(blockSize);
    int    size = length * sizeof(float);
    float* sum  = (float*)malloc(size);

    // 开辟显存空间
    float *sumGPU, *aGPU, *bGPU;
    hipMalloc((void**)&sumGPU, size);
    hipMalloc((void**)&aGPU, size);
    hipMalloc((void**)&bGPU, size);

    // 内存->显存
    hipMemcpy((void*)aGPU, (void*)a, size, hipMemcpyHostToDevice);
    hipMemcpy((void*)bGPU, (void*)b, size, hipMemcpyHostToDevice);

    // 运算
    matAdd_cuda<float><<<block, thread>>>(aGPU, bGPU, sumGPU);
    hipDeviceSynchronize();

    // 显存->内存
    hipMemcpy(sum, sumGPU, size, hipMemcpyDeviceToHost);

    // 释放显存
    hipFree(sumGPU);
    hipFree(aGPU);
    hipFree(bGPU);

    return sum;
}

int main(int argc, char** argv)
{
    // 创建数组
    const int length = 10;
    float     a[length], b[length];
    for (int i = 0; i < length; i++)
    {
        a[i] = 1;
        b[i] = 2;
    }

    float* c = matAdd(a, b, length);

    // 输出查看是否完成计算
    for (int i = 0; i < length; i++)
    {
        std::cout << a[i] << " " << b[i] << " " << c[i] << std::endl;
    }

    return EXIT_SUCCESS;
}
