#include "hip/hip_runtime.h"
// c
#include <cfloat>

// cpp
#include <chrono>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <fstream>
#include <iostream>
#include <memory>
#include <string>

// cuda
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"

// user
#include "raytracinginoneweekendincuda/box.h"
#include "raytracinginoneweekendincuda/bvh.h"
#include "raytracinginoneweekendincuda/camera.h"
#include "raytracinginoneweekendincuda/hitable.h"
#include "raytracinginoneweekendincuda/hitable_list.h"
#include "raytracinginoneweekendincuda/material.h"
#include "raytracinginoneweekendincuda/moving_sphere.h"
#include "raytracinginoneweekendincuda/perlin.h"
#include "raytracinginoneweekendincuda/ray.h"
#include "raytracinginoneweekendincuda/rectangle.h"
#include "raytracinginoneweekendincuda/sphere.h"
#include "raytracinginoneweekendincuda/texture.h"
#include "raytracinginoneweekendincuda/util.h"
#include "raytracinginoneweekendincuda/vec3.h"
#include "raytracinginoneweekendincuda/volume.h"

// 3rdparty
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

const std::string IMAGE_FILE  = "E:/Study/CodeProj/RT-in-One-Weekend-Series/asset/world.jpg";
const std::string OUTPUT_FILE = "output.ppm";

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":"
                  << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, hitable** world, hiprandState* local_rand_state)
{
    ray  cur_ray         = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++)
    {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec))
        {
            ray  scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state))
            {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else { return vec3(0.0, 0.0, 0.0); }
        }
        else
        {
            vec3  unit_direction = unit_vector(cur_ray.direction());
            float t              = 0.5f * (unit_direction.y() + 1.0f);
            vec3  c              = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0);  // exceeded recursion
}

__device__ vec3 ray_color(const ray&   r,
                          hitable**    world,
                          hiprandState* local_rand_state,
                          const vec3&  background = {0})
{
    ray  cur_ray         = r;
    vec3 cur_attenuation = vec3(1);
    for (int i = 0; i < 50; i++)
    {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec))
        {
            ray  scattered;
            vec3 attenuation;

            vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            if (dot(cur_ray.direction(), rec.normal) > 0) { emitted = vec3{0}; }

            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state))
            {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else
            {
                // 自发光材质，没有scatter
                return cur_attenuation * emitted;
            }
        }
        else
        {
            // sky light
            vec3  unit_direction = unit_vector(cur_ray.direction());
            float t              = 0.5f * (unit_direction.y() + 1.0f);
            vec3  sky_light      = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);

            // return cur_attenuation * (background + sky_light * 0.05);
            return cur_attenuation * background;
        }
    }
    return vec3(0);  // exceeded recursion
}

__global__ void rand_init(hiprandState* rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) { hiprand_init(1984, 0, 0, rand_state); }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3*        fb,
                       int          max_x,
                       int          max_y,
                       int          ns,
                       camera**     cam,
                       hitable**    world,
                       hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int         pixel_index      = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3        col(0);
    for (int s = 0; s < ns; s++)
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray   r = (*cam)->get_ray(u, v, &local_rand_state);
        // col += color(r, world, &local_rand_state);
        col += ray_color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    fb[pixel_index] = col.gamma_correction();
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable**      d_list,
                             hitable**      d_world,
                             camera**       d_camera,
                             int            nx,
                             int            ny,
                             int            n_objects,
                             unsigned char* texture_device_data,
                             int            texture_width,
                             int            texture_height,
                             hiprandState*   rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        hiprandState local_rand_state = *rand_state;

        // texture
        auto red_mat   = new lambertian(new const_texture(vec3(0.65, 0.05, 0.05)));
        auto white_mat = new lambertian(new const_texture(vec3(0.73, 0.73, 0.73)));
        auto green_mat = new lambertian(new const_texture(vec3(0.12, 0.45, 0.15)));
        auto light_mat = new diffuse_light(new const_texture(vec3(15.0)));

        // main objects
        int i       = 0;
        d_list[i++] = new yz_rect(0, 555, 0, 555, 555, green_mat);
        d_list[i++] = new yz_rect(0, 555, 0, 555, 0, red_mat);
        d_list[i++] = new xz_rect(0, 555, 0, 555, 0, white_mat);
        d_list[i++] = new xy_rect(0, 555, 0, 555, 555, white_mat);
        d_list[i++] = new xz_rect(0, 555, 0, 555, 555, white_mat);
        // 2 box
        auto* dieletric = new sphere(vec3(210, 100, 150), 80, new dielectric(1.5));
        auto* volume =
            new constant_medium(dieletric, 0.02, new const_texture(vec3(0, 0, 0.8)), rand_state);
        d_list[i++] = new SSS_volume(dieletric, volume);
        d_list[i++] =
            new rotate_y(new box(vec3(265, 0, 295), vec3(430, 330, 460), white_mat), -45.0f);

        // light
        d_list[i++] = new flip_face(new xz_rect(213, 343, 227, 332, 554, light_mat));

        // create the world
        *d_world = new hitable_list(d_list, n_objects);

        // camera
        const auto aspect_ratio = double(nx) / ny;
        vec3       lookfrom(278, 278, -800);
        vec3       lookat(278, 278, 0);
        vec3       vup(0, 1, 0);
        auto       dist_to_focus = 10.0;
        auto       aperture      = 0.0;
        auto       vfov          = 40.0;
        float      time0 = 0.0, time1 = 1.0;
        *d_camera = new camera(lookfrom, lookat, vup, vfov, aspect_ratio, aperture, dist_to_focus,
                               time0, time1);
    }
}

__global__ void free_world(hitable** d_list, int num_objects, hitable** d_world, camera** d_camera)
{
    for (int i = 0; i < num_objects; i++)
    {
        // mat_ptr 已经在析构函数上释放了
        // FIXME - 并非所有的hitable都是sphere类
        delete d_list[i];
        // FIXME - delete好像不会检查指针的有效性，直接全删了，
        // 我只有6个对象，delete d_list[20]全删了
    }
    delete *d_world;
    delete *d_camera;
}

int main()
{
    const int nx = 800;
    const int ny = 800;
    const int ns = 128;
    const int tx = 8;
    const int ty = 8;

    std::cout << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cout << "in " << tx << "x" << ty << " blocks.\n";

    constexpr int    num_pixels = nx * ny;
    constexpr size_t fb_size    = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *fb, *fb_host;
    fb_host = new vec3[num_pixels];
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // import image to cuda
    int            width, height, channels;
    unsigned char* texture_host_data = stbi_load(IMAGE_FILE.c_str(), &width, &height, &channels, 0);
    int            texture_size      = width * height * channels;
    unsigned char* texture_device_data;
    checkCudaErrors(hipMallocManaged((void**)&texture_device_data, texture_size));
    checkCudaErrors(
        hipMemcpy(texture_device_data, texture_host_data, texture_size, hipMemcpyHostToDevice));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // NOTE - num of objects
    const int num_objects = 8;

    // make our world of hitables & the camera
    hitable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, num_objects * sizeof(hitable*)));
    hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, num_objects, texture_device_data,
                           width, height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto start = std::chrono::system_clock::now();
    // Render our buffer
    const dim3 blocks(nx / tx + 1, ny / ty + 1);
    const dim3 threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    auto stop     = std::chrono::system_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "Time:  " << duration.count() / 1000.0f << " s\n";

    // transfer FB to host
    checkCudaErrors(hipMemcpy(fb_host, fb, fb_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    //  Output FB as Image
    std::ofstream out(OUTPUT_FILE);
    if (!out.is_open())
    {
        std::cerr << "Failed to open file: " << OUTPUT_FILE << std::endl;
        return EXIT_FAILURE;
    }
    out << "P3\n" << nx << ' ' << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--)
    {
        for (int i = 0; i < nx; i++)
        {
            size_t pixel_index = j * nx + i;
            vec3   col         = fb_host[pixel_index] * 255.999;
            out << col << "\n";
        }
    }
    out.close();

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, num_objects, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));
    delete[] texture_host_data, fb_host;
    checkCudaErrors(hipFree(texture_device_data));

    hipDeviceReset();

    return EXIT_SUCCESS;
}
